
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

__global__ void vector_add(int *a, int *b, int *c, int N)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
	  c[index] = a[index] + b[index];
}

#define THREADS_PER_BLOCK 512

int main(int argc, char*argv[])
{
    int N = atoi(argv[1]);
    assert(N>0 && N<=1000000);
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	vector_add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c, N );


	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


	printf( "c[0] = %d\n",0,c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
